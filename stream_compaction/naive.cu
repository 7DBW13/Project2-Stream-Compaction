#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

// Block size used for CUDA kernel launch
#define BLOCK_SIZE 128

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        // Add each value at (index-2^(d-1)) to the value at (index)
        __global__ void kern_add_pairs(int n, int d, const int* idata, int* odata) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n || index < (1 << (d - 1))) {
                return;
            }

            odata[index] = idata[index] + idata[index - (1 << (d - 1))];
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // Create device array and a buffer
            int *dev_array;
            int *dev_array_buf;
            hipMalloc((void **)&dev_array, n * sizeof(int));
            hipMalloc((void **)&dev_array_buf, n * sizeof(int));
            checkCUDAError("hipMalloc failed!");

            // Copy data to GPU
            hipMemcpy(dev_array, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            hipMemcpy(dev_array_buf, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy failed!");

            dim3 fullBlocksPerGrid((n + BLOCK_SIZE - 1) / BLOCK_SIZE);

            timer().startGpuTimer();
            
            // Add for log(n) times
            for (int d = 1; d <= ilog2ceil(n); d++) {
                kern_add_pairs << <fullBlocksPerGrid, BLOCK_SIZE >> > (n, d, dev_array, dev_array_buf);
                checkCUDAError("kern_add_pairs failed!");

                // Ping-pong the buffers
                hipMemcpy(dev_array, dev_array_buf, sizeof(int) * n, hipMemcpyDeviceToDevice);
                checkCUDAError("ping-pong failed!");
            }

            // Set identity
            odata[0] = 0;

            timer().endGpuTimer();

            // Copy data back
            // Shift inclusive scan to exclusive scan           
            hipMemcpy(odata + 1, dev_array, sizeof(int) * (n - 1), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy back failed!");

            // Cleanup
            hipFree(dev_array);
            hipFree(dev_array_buf);
            checkCUDAError("hipFree failed!");
        }
    }
}
